#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""

#include <stdio.h>

#define blocksize 64
#define BLOCK blocksize
#define N (1 << 10)

__global__ void matrixs_GPU(float* matrix_A_dev, float* matrix_B_dev, float* matrix_C_dev);
void print(float* matrix_A);

int main()
{
    float* matrix_A = (float*)malloc(sizeof(float) * N * N);
    float* matrix_B = (float*)malloc(sizeof(float) * N * N);
    float* matrix_C = (float*)malloc(sizeof(float) * N * N);

    srand(time(0));

    // random initialize A,B and initialize C with 0
    for (int i = 0; i < N * N; i++) {
        matrix_A[i] = 150 + rand() % 120;
        matrix_B[i] = 150 + rand() % 120;
        matrix_C[i] = 0;
    }

    float* matrix_A_dev;
    float* matrix_B_dev;
    float* matrix_C_dev;

    hipMalloc((void**)&matrix_A_dev, N * N * sizeof(float));
    hipMalloc((void**)&matrix_B_dev, N * N * sizeof(float));
    hipMalloc((void**)&matrix_C_dev, N * N * sizeof(float));
    hipMemcpy(matrix_A_dev, matrix_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(matrix_B_dev, matrix_B, N * N * sizeof(float), hipMemcpyHostToDevice);


    dim3 block(blocksize, blocksize);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    matrixs_GPU << < grid, block >> > (matrix_A_dev, matrix_B_dev, matrix_C_dev);
    hipMemcpy(matrix_C, matrix_C_dev, N * N * sizeof(float), hipMemcpyDeviceToHost);


   /* print(matrix_C);
    gemm_verify(matrix_A, matrix_B, matrix_C);
    print(matrix_C);*/

    free(matrix_A);
    free(matrix_B);
    free(matrix_C);
    hipFree(matrix_A_dev);
    hipFree(matrix_B_dev);
    hipFree(matrix_C_dev);
}
__global__ void matrixs_GPU(float* matrix_A_dev, float* matrix_B_dev, float* matrix_C_dev) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;


    float res = 0.0;


    for (int i = 0; i < (int)(ceil((float)N / blockDim.x)); i++) {
        __shared__ float sharedA[BLOCK][BLOCK];
        __shared__ float sharedB[BLOCK][BLOCK];

        if (i * BLOCK + tx < N && row < N) {
            sharedA[ty][tx] = matrix_A_dev[row * N + i * blockDim.x + tx];
        }
        else
            sharedA[ty][tx] = 0.0;

        if (i * BLOCK + ty < N && col < N) {
            sharedB[ty][tx] = matrix_B_dev[(i * BLOCK + ty) * N + col];
        }
        else
            sharedB[ty][tx] = 0.0;

        __syncthreads();

        for (int j = 0; j < N; j++) {
            res += sharedA[ty][j] * sharedB[j][tx];
        }

        __syncthreads();
    }

    if (row < N && col < N)
        matrix_C_dev[row * N + col] = res;
}



void gemm_verify(float* matrix_A, float* matrix_B, float* matrix_C) {
    int i, j, k;
    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            matrix_C[i * N + j] = 0;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            for (k = 0; k < N; k++) {
                matrix_C[i * N + j] += matrix_A[i * N + k] * matrix_B[k * N + j];
            }
        }
    }
}

void print(float* matrix_A) {
    int i, j;
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            printf("%f ", matrix_A[i * N + j]);
        }
        printf("\n");
    }
}


